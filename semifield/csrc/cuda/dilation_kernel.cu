#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__global__ void dilation_cuda_forward_kernel(
    const scalar_t* a,
    const scalar_t* b,
    scalar_t* result,
    int64_t size) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    result[idx] = a[idx] * b[idx];
  }
}

template <typename scalar_t>
__global__ void dilation_cuda_backward_kernel(
    const scalar_t* grad_output,
    const scalar_t* a,
    const scalar_t* b,
    scalar_t* grad_a,
    scalar_t* grad_b,
    const int64_t size) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_a[idx] = grad_output[idx] * b[idx];
        grad_b[idx] = grad_output[idx] * a[idx];
    }
}

at::Tensor dilation_cuda_forward(const at::Tensor& a, const at::Tensor& b) {
    auto result = torch::empty_like(a);
    const int threads = 1024;
    const int blocks = (a.numel() + threads - 1) / threads;
  
    AT_DISPATCH_FLOATING_TYPES(a.scalar_type(), "dilation_forward_cuda", ([&] {
      dilation_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        a.data_ptr<scalar_t>(),
        b.data_ptr<scalar_t>(),
        result.data_ptr<scalar_t>(),
        a.numel());
    }));
  
    return result;
  }
  
  std::vector<at::Tensor> dilation_cuda_backward(const at::Tensor& grad_output, const at::Tensor& a, const at::Tensor& b) {
    auto grad_a = torch::empty_like(a);
    auto grad_b = torch::empty_like(b);
    const int threads = 1024;
    const int blocks = (a.numel() + threads - 1) / threads;
  
    AT_DISPATCH_FLOATING_TYPES(a.scalar_type(), "dilation_backward_cuda", ([&] {
      dilation_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        grad_output.data_ptr<scalar_t>(),
        a.data_ptr<scalar_t>(),
        b.data_ptr<scalar_t>(),
        grad_a.data_ptr<scalar_t>(),
        grad_b.data_ptr<scalar_t>(),
        a.numel());
    }));
  
    return {grad_a, grad_b};
  }